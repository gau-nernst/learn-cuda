
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdio.h>
#include <assert.h>
#include <cstdint>
#include <hip/hip_bf16.h>

#define PRINT_IF(cond, ...) if (cond) printf(__VA_ARGS__);

__host__ __device__ constexpr int cdiv(int a, int b) { return (a + b - 1) / b; }
constexpr bool is_power_of_two(int x) { return x > 0 && (x & (x - 1)) == 0; }  // https://stackoverflow.com/a/1804686
constexpr int WARP_SIZE = 32;


template <int BLOCK_SIZE, int HEIGHT, int WIDTH, typename T>
__device__ void load_shared_b128(const T *in, int in_row_stride, T *out, int tid) {
  // number of elements to do 128-bit load
  // e.g. FP32 -> 4 elements, BF16 -> 8 elements.
  using load_type = uint4;
  constexpr int num_elems = sizeof(load_type) / sizeof(T);

  for (int idx = tid * num_elems; idx < HEIGHT * WIDTH; idx += BLOCK_SIZE * num_elems) {
    const int row = idx / WIDTH;
    const int col = idx % WIDTH;
    load_type tmp = reinterpret_cast<const load_type *>(&in[row * in_row_stride + col])[0];
    reinterpret_cast<load_type *>(&out[row * WIDTH + col])[0] = tmp;
  }
}

__device__ uint32_t cvta_shared(void const *ptr) { return static_cast<uint32_t>(__cvta_generic_to_shared(ptr)); }

template <typename T> __device__ void mma_m16n8k8(uint32_t A[2], uint32_t B, float acc[4]);
template <> __device__ void mma_m16n8k8<half>(uint32_t A[2], uint32_t B, float acc[4]) {
  asm volatile (
    "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32 "
    "{%0, %1, %2, %3}, "  // D
    "{%4, %5}, "          // A
    "{%6}, "              // B
    "{%7, %8, %9, %10};"  // C
    : "=f"(acc[0]), "=f"(acc[1]), "=f"(acc[2]), "=f"(acc[3])
    : "r"(A[0]), "r"(A[1]),
      "r"(B),
      "f"(acc[0]), "f"(acc[1]), "f"(acc[2]), "f"(acc[3])
  );
}
template <> __device__ void mma_m16n8k8<hip_bfloat16>(uint32_t A[2], uint32_t B, float acc[4]) {
  asm volatile (
    "mma.sync.aligned.m16n8k8.row.col.f32.bf16.bf16.f32 "
    "{%0, %1, %2, %3}, "  // D
    "{%4, %5}, "          // A
    "{%6}, "              // B
    "{%7, %8, %9, %10};"  // C
    : "=f"(acc[0]), "=f"(acc[1]), "=f"(acc[2]), "=f"(acc[3])
    : "r"(A[0]), "r"(A[1]),
      "r"(B),
      "f"(acc[0]), "f"(acc[1]), "f"(acc[2]), "f"(acc[3])
  );
}

template <typename T> __device__ ushort f32_to_b16(float x);
template <> __device__ ushort f32_to_b16<half>(float x) { return __half_as_ushort(__float2half(x)); }
template <> __device__ ushort f32_to_b16<hip_bfloat16>(float x) { return __bfloat16_as_ushort(__float2bfloat16(x)); }

template <
  int BLOCK_M, int BLOCK_N, int BLOCK_K,
  int WARP_M, int WARP_N, int WARP_K,
  typename T>
__global__ void matmul_v1_kernel(const T *A, const T *B, T *C, int M, int N, int K) {
  constexpr int MMA_M = 16, MMA_N = 8, MMA_K = 8;
  static_assert(BLOCK_M % WARP_M == 0);
  static_assert(BLOCK_N % WARP_N == 0);
  static_assert(BLOCK_K % WARP_K == 0);
  static_assert(WARP_M % MMA_M == 0);
  static_assert(WARP_N % MMA_N == 0);
  static_assert(WARP_K % MMA_K == 0);
  constexpr int BLOCK_SIZE = (BLOCK_M * BLOCK_N) / (WARP_M * WARP_N) * WARP_SIZE;
  constexpr int NUM_MMA_M = WARP_M / MMA_M;
  constexpr int NUM_MMA_N = WARP_N / MMA_N;
  constexpr int NUM_MMA_K = WARP_K / MMA_K;

  const int tid = threadIdx.x;
  const int block_id = blockIdx.x;
  const int warp_id = tid / WARP_SIZE;
  const int lane_id = tid % WARP_SIZE;

  const int num_blocks_per_row = cdiv(N, BLOCK_N);
  const int block_id_m = block_id / num_blocks_per_row;
  const int block_id_n = block_id % num_blocks_per_row;
  const int offset_m = block_id_m * BLOCK_M;
  const int offset_n = block_id_n * BLOCK_N;

  constexpr int num_warps_per_row = BLOCK_N / WARP_N;
  const int warp_id_m = warp_id / num_warps_per_row;
  const int warp_id_n = warp_id % num_warps_per_row;
  const int warp_tile_offset_m = warp_id_m * WARP_M;
  const int warp_tile_offset_n = warp_id_n * WARP_N;

  // A is row-major, B is column-major
  A += offset_m * K;
  B += offset_n * K;

  __shared__ T A_shared[BLOCK_M * BLOCK_K];
  __shared__ T B_shared[BLOCK_N * BLOCK_K];

  float acc[NUM_MMA_M][NUM_MMA_N][4] = {0.0f};  // each thread holds 4 output float
  uint32_t A_reg[NUM_MMA_M][NUM_MMA_K][2];      // each thread holds 2 input f16x2
  uint32_t B_reg[NUM_MMA_N][NUM_MMA_K];         // each thread holds 1 input f16x1

  // first A and B warp-tile along BLOCK_K dim (we will iterate along BLOCK_K with step_size=WARP_K)
  const T *A_warp_tile = reinterpret_cast<const T *>(A_shared) + warp_tile_offset_m * BLOCK_K;
  const T *B_warp_tile = reinterpret_cast<const T *>(B_shared) + warp_tile_offset_n * BLOCK_K;

  for (int block_k = 0; block_k < K; block_k += BLOCK_K) {
    load_shared_b128<BLOCK_SIZE, BLOCK_M, BLOCK_K>(A, K, A_shared, tid);
    load_shared_b128<BLOCK_SIZE, BLOCK_N, BLOCK_K>(B, K, B_shared, tid);
    __syncthreads();

    for (int warp_k = 0; warp_k < BLOCK_K; warp_k += WARP_K) {
      // load data from shared memory to registers using ldmatrix
      // https://docs.nvidia.com/cuda/parallel-thread-execution/#warp-level-matrix-instructions-ldmatrix

      // convert generic address to .shared state space address expected by inline PTX
      // thread 0 holds address of row 0
      // thread 1 holds address of row 1, and so on
      uint32_t A_tile_addr = cvta_shared(A_warp_tile + lane_id * BLOCK_K + warp_k);
      uint32_t B_tile_addr = cvta_shared(B_warp_tile + lane_id * BLOCK_K + warp_k);

      // load A to registers
      // ldmatrix can only load 8x8 matrix. for 16x8 tile, we need to use x2
      for (int mma_tile_id_m = 0; mma_tile_id_m < NUM_MMA_M; mma_tile_id_m++) {
        for (int mma_tile_id_k = 0; mma_tile_id_k < NUM_MMA_K; mma_tile_id_k++) {
          uint32_t *A_reg_frag = A_reg[mma_tile_id_m][mma_tile_id_k];
          uint32_t A_local = A_tile_addr + (mma_tile_id_m * MMA_M * BLOCK_K + mma_tile_id_k * MMA_K) * sizeof(T);
          asm volatile (
            "ldmatrix.sync.aligned.m8n8.x2.shared.b16 {%0, %1}, [%2];"
            : "=r"(A_reg_frag[0]), "=r"(A_reg_frag[1])  // output
            : "r"(A_local)  // input
          );
        }
      }

      // load B to registers
      for (int mma_tile_id_n = 0; mma_tile_id_n < NUM_MMA_N; mma_tile_id_n++) {
        for (int mma_tile_id_k = 0; mma_tile_id_k < NUM_MMA_K; mma_tile_id_k++) {
          uint32_t B_local = B_tile_addr + (mma_tile_id_n * MMA_N * BLOCK_K + mma_tile_id_k * MMA_K) * sizeof(T);
          asm volatile (
            "ldmatrix.sync.aligned.m8n8.x1.shared.b16 {%0}, [%1];"
            : "=r"(B_reg[mma_tile_id_n][mma_tile_id_k]) // output
            : "r"(B_local)  // input
          );
        }
      }

      // call mma
      // https://docs.nvidia.com/cuda/parallel-thread-execution/#warp-level-matrix-fragment-mma-1688
      for (int mma_tile_id_m = 0; mma_tile_id_m < NUM_MMA_M; mma_tile_id_m++)
        for (int mma_tile_id_n = 0; mma_tile_id_n < NUM_MMA_N; mma_tile_id_n++)
          for (int mma_tile_id_k = 0; mma_tile_id_k < NUM_MMA_K; mma_tile_id_k++)
            mma_m16n8k8<T>(
              A_reg[mma_tile_id_m][mma_tile_id_k],
              B_reg[mma_tile_id_n][mma_tile_id_k],
              acc[mma_tile_id_m][mma_tile_id_n]);
    }
    __syncthreads();

    A += BLOCK_K;
    B += BLOCK_K;
  }

  const int C_offset_m = offset_m + warp_tile_offset_m;
  const int C_offset_n = offset_n + warp_tile_offset_n;
  C += C_offset_m * N + C_offset_n;

  // check output layout here
  // https://docs.nvidia.com/cuda/parallel-thread-execution/#warp-level-matrix-fragment-mma-1688
  const int a0_row = lane_id >> 2; 
  const int a0_col = (lane_id % 4) * 2;
  C += a0_row * N + a0_col;

  for (int mma_tile_id_m = 0; mma_tile_id_m < NUM_MMA_M; mma_tile_id_m++)
    for (int mma_tile_id_n = 0; mma_tile_id_n < NUM_MMA_N; mma_tile_id_n++) {
      T *C_local = C + mma_tile_id_m * MMA_M * N + mma_tile_id_n * MMA_N;
      float *acc_frag = acc[mma_tile_id_m][mma_tile_id_n];
      ushort2 tmp;

      // write a0 and a1
      tmp.x = f32_to_b16<T>(acc_frag[0]);
      tmp.y = f32_to_b16<T>(acc_frag[1]);
      reinterpret_cast<ushort2 *>(C_local)[0] = tmp;

      // write a2 and a3
      tmp.x = f32_to_b16<T>(acc_frag[2]);
      tmp.y = f32_to_b16<T>(acc_frag[3]);
      reinterpret_cast<ushort2 *>(C_local + 8 * N)[0] = tmp;
    }
}

void matmul_v1(const hip_bfloat16 *A, const hip_bfloat16 *B, hip_bfloat16 *C, int M, int N, int K) {
  assert(is_power_of_two(M) && "M must be a power of 2");
  assert(is_power_of_two(N) && "N must be a power of 2");
  assert(is_power_of_two(K) && "K must be a power of 2");

  const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 32;
  const int WARP_M = 64, WARP_N = 32, WARP_K = 16;

  const int BLOCK_SIZE = (BLOCK_M * BLOCK_N) / (WARP_M * WARP_N) * WARP_SIZE;
  const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
  matmul_v1_kernel<
    BLOCK_M, BLOCK_N, BLOCK_K,
    WARP_M, WARP_N, WARP_K><<<grid_size, BLOCK_SIZE>>>(A, B, C, M, N, K);
}
